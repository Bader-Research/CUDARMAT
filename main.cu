#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <xmalloc.cuh>
#include <sys/time.h>
#include <stdint.h>
#include <getopt.h>
#include <errno.h>
#include <util.h>

#define DEBUG 0

//////////////////////////////
#define INITRANDMULT       0x015A4E35
#define INITRANDINCREMENT  997
#define RANDMULT           214013
#define RANDINCREMENT      2531011
#define TESTRAND           100
#define MAXBLOCKSIZE       512
#define EDGEBATCHSIZE		134217728
#define ACTIONBATCHSIZE		16777216

// Errors
//////////////////////////////
typedef enum  {
   NO_ERROR,
   DELETIONS_OVERRAN_INSERTIONS               
} graphError_t;

const char * graphErrorString[] = {
   "NO ERROR",
   "DELETIONS OVERRAN INSERTIONS - RARE RANDOM ERROR, RUN IT AGAIN."
};


// Device Function Prototypes
//////////////////////////////
__device__ uint32_t cudaRand(uint32_t * randVal);
__global__ void cudaRMATEdges(uint32_t * randVals, uint32_t SCALE, uint32_t edgesPerThread, uint32_t numthreads, 
                               float pA, float pB, float pC, float pD, uint32_t * edgeArray);
__global__ void cudaGenerateActions(uint32_t * randVals, uint32_t edges, uint32_t actions, uint32_t numthreads, 
                                    float pDelete, uint32_t * edgeArray, uint32_t * generatedEdges, int32_t * actionsEdgeArray, graphError_t * error); 

#if(DEBUG)
__global__ void cudaDebugRand(uint32_t * randOut, uint32_t * randVals);
__global__ void cudaDebugEdgeList(uint32_t vertices, uint32_t * edgeList, uint32_t size);
#endif

// Host Functions Prototypes
/////////////////////////////
__host__ void hostParseArgs(int argc, char** argv);
__host__ void hostInitCudaRand(); 
__host__ void hostRMATandFileIO();
__host__ int  hostCompareEdges(const void * a, const void * b);   
__host__ void hostFreeCudaRand();

#if (DEBUG)
__host__ void hostDebugTestRand();
#endif

// Global Device Variables
/////////////////////////////
uint32_t       * d_uip_randvals;
uint32_t       * d_uip_edgelist;
uint32_t			* h_uip_edgelist;
int32_t        * d_ip_actionslist;
int32_t        * h_ip_actionslist;
void           * d_vp_cudastinger;
void           * d_vp_cudavertices;
graphError_t   * d_gep_error;

// Global Host Variables
////////////////////////////
uint32_t h_ui_scale = 14;
uint32_t h_ui_edgefactor = 16;
uint32_t h_ui_actions = 512 * 512 * 10;
uint32_t h_ui_threads = 512;
uint32_t h_ui_blocks = 512;
uint32_t h_ui_vertices = 4096;
uint32_t h_ui_edges = 32768;
const char * h_s_infile = NULL;
const char * h_s_outfile = NULL;
const char * h_s_dimacsoutfile = NULL;
const char * h_s_stinger_outfile = NULL;
const char * h_s_stinger_actionsfile = NULL;
int h_i_streaming = 0;
                                                                                                  
// Host Functions
/////////////////////////////
__host__ int main(int argc, char** argv) {
   printf("CUDA CP2 Implementation\n");
   hostParseArgs(argc, argv);

   d_gep_error = (graphError_t *)cudaXmalloc(sizeof(graphError_t));
   graphError_t hosterror = NO_ERROR;
   hipMemcpy(d_gep_error, &hosterror, sizeof(graphError_t), hipMemcpyHostToDevice);

   if(h_ui_threads % 8 != 0 || h_ui_blocks % 8 != 0) {
      fprintf(stderr, "ERROR: Blocks and Threads must be multiples of 8\n");
      exit(-1);
   }

   tic_reset();
   hostInitCudaRand();

#if(DEBUG)
   hostDebugTestRand();
#endif

   hostRMATandFileIO();

   hipMemcpy(&hosterror, d_gep_error, sizeof(graphError_t), hipMemcpyDeviceToHost);

   hostFreeCudaRand();
   hipFree(d_gep_error);

   hipDeviceSynchronize();
   printf("\nfree() %f", tic_sincelast());
   printf("\nTotalTime %f\n", tic_total());

	if(hipPeekAtLastError() != hipSuccess) 
		printf("**********************************"
		"\nCUDA ERROR OCCURED :\n\t%s\nRESULTS MAY NOT BE VALID\n"
		"**********************************\n", hipGetErrorString(hipGetLastError()));
   else if(hosterror != NO_ERROR)
      printf("************************"
      "\nGRAPH ERROR OCCURED:"
      "\n%d - %s"
       "\n************************\n", hosterror, graphErrorString[hosterror]);
   else
      printf("NO ERRORS\n");

   return 0;
}

__host__ void hostParseArgs(int argc, char** argv) {
   static struct option long_options[] = {
      {"scale", required_argument, 0, 's'},
      {"edgefactor", required_argument, 0, 'e'},
      {"actions", required_argument, 0, 'a'},
      {"help", no_argument, 0, 'h'},
      {"blocks", required_argument, 0, 'b'},
      {"threads", required_argument, 0, 't'},
      {"outfile", required_argument, 0, 'o'},
      {"dimacsoutfile", required_argument, 0, 'd'},
      {"STINGERoutputfile", required_argument, 0, 'S'},
      {"STINGERactionsfile", required_argument, 0, 'A'},
      {"CUDADevice", required_argument, 0, 'c'},
      {0, 0, 0, 0}
   };

   int32_t intout;

   while(1) {
      int option_index = 0;                          
      int c = getopt_long(argc, argv, "s:e:a:h?b:t:o:d:S:A:c:", long_options, &option_index);
      extern char * optarg;
      extern int    optind, opterr, optopt;

      if(-1 == c)
         break;
      
      switch(c) {
         default:
            printf("Unrecognized option: %c\n\n", c);
         case '?':
         case 'h':
            printf("\nUsage"
                   "\n====="
                   "\n\t-s --scale=SCALE"
                   "\n\t-e --edgefact=EDGEFACT"
                   "\n\t-a --actions=NUMBEROFACTIONS"
                   "\n\t-o --outfile=OUTPUTEDGELISTFILE"
                   "\n\t-d --dimacsoutfile=DIMACSFORMATOUTPUTFILE"
                   "\n\t-S --STINGERoutputfile=STINGEROUTPUTFILE"
						 "\n\t-A --STINGERactionsfile=STINGERACTIONSFILE"
                   "\n\n\tTUNING"
                   "\n\t-b --blocks=BLOCKS"
                   "\n\t-t --threads=THREADS"
                   "\n\t-d --CUDADevice=DEVICENUMBER - if not specified, default is used"
                   "\n\nEdge list files are binary files containing uint32 scale, edgefactor, and"
                   " edges as ordered pairs of uint32\n");
            exit(0);
            break;
         case 's':
            errno = 0;
            intout = strtol(optarg, NULL, 10);
            if(errno || intout < 0) {
               printf("Error - Scale = %s\n", optarg);
               exit(-1);
            }
            h_ui_scale = intout;
            break;
         case 'e':
            errno = 0;
            intout = strtol(optarg, NULL, 10);
            if(errno || intout < 0) {
               printf("Error - Edgefactor = %s\n", optarg);
               exit(-1);
            }
            h_ui_edgefactor = intout;
            break;
         case 'a':
            errno = 0;
            intout = strtol(optarg, NULL, 10);
            if(errno || intout < 0) {
               printf("Error - Actions = %s\n", optarg);
               exit(-1);
            }
            h_ui_actions = intout;
            break;
         case 'b':
            errno = 0;
            intout = strtol(optarg, NULL, 10);
            if(errno || intout < 0) {
               printf("Error - BLOCKS = %s\n", optarg);
               exit(-1);
            }
            h_ui_blocks = intout; 
            break;
         case 't':
            errno = 0;
            intout = strtol(optarg, NULL, 10);
            if(errno || intout < 0) {
               printf("Error - THREADS = %s\n", optarg);
               exit(-1);
            }
            h_ui_threads = intout; 
            break;
         case 'c':
            errno =0;
            intout = strtol(optarg, NULL, 10);
            if(errno || intout < 0) {
               printf("Error - CUDA Device = %s\n", optarg);
               exit(-1);
            }
            hipSetDevice(intout);
            break;
         case 'i':
            if(optarg != NULL)
               h_s_infile = optarg;
            break;
         case 'o':
            if(optarg != NULL)
               h_s_outfile = optarg;
            break; 
         case 'd':
            if(optarg != NULL)
               h_s_dimacsoutfile = optarg;
            break;
         case 'S':
            if(optarg != NULL)
               h_s_stinger_outfile = optarg;
            break;
         case 'A':
            if(optarg != NULL)
               h_s_stinger_actionsfile = optarg;
            break;
         case 'p':
            h_i_streaming = 1;
            break;
      }
   }
   
   h_ui_vertices = (1L << h_ui_scale);
   h_ui_edges = h_ui_vertices * h_ui_edgefactor;
   
   if(h_s_infile == NULL) {
      printf("<BLOCKS, THREADS>  <%u, %u>\n", h_ui_blocks, h_ui_threads);
      printf("\n\tScale      %d\n\tEdgefactor %d\n\tActions    %d\n\t<V,E>      <%d,%d>\n", 
         h_ui_scale, h_ui_edgefactor, h_ui_actions, h_ui_vertices, h_ui_edges);
   }
}

__host__ void hostInitCudaRand() {
   uint32_t totalThreads = h_ui_blocks * h_ui_threads;
   uint32_t * hostRandVals = (uint32_t *)xmalloc(totalThreads * sizeof(uint32_t));
   d_uip_randvals = (uint32_t *)cudaXmalloc(totalThreads * sizeof(uint32_t));

   struct timeval tv;
   gettimeofday(&tv, NULL);
   hostRandVals[0] = tv.tv_sec * INITRANDMULT + INITRANDINCREMENT;
   uint32_t i;
   for(i = 1; i < totalThreads; ++i) {
      hostRandVals[i] = hostRandVals[i-1] * INITRANDMULT + INITRANDINCREMENT;
   }

   hipMemcpy(d_uip_randvals, hostRandVals, totalThreads * sizeof(uint32_t), hipMemcpyHostToDevice);

   free(hostRandVals);
   
   hipDeviceSynchronize();
   printf("\nhostInitCudaRand() %f", tic_sincelast());
}

__host__ void hostRMATandFileIO() {
	if(h_ui_edges % (h_ui_blocks * h_ui_threads) != 0) {
		printf("ERROR: Edges must divide evenly by blocks * threads.\n");
		exit(-1);
	};

	d_uip_edgelist    		= (uint32_t *)cudaXmalloc(EDGEBATCHSIZE * 4 * sizeof(uint32_t));
	h_uip_edgelist				= (uint32_t *)xmalloc(h_ui_edges * 4 * sizeof(uint32_t));
	d_ip_actionslist        = (int32_t  *)cudaXmalloc(ACTIONBATCHSIZE * 4 * sizeof(int32_t));
	uint32_t * tempactions  = (uint32_t *)cudaXmalloc(ACTIONBATCHSIZE * 4 * sizeof(uint32_t));
	h_ip_actionslist			= (int32_t  *)xmalloc(h_ui_actions * 4 * sizeof(int32_t));

	hipDeviceSynchronize();
	printf("\nedgeListMalloc %f", tic_sincelast());

	if(h_ui_actions % (h_ui_blocks * h_ui_threads) != 0) {
		printf("ERROR: Actions must divide evenly by blocks * threads.\n");
		exit(-1);
	}

	uint32_t * edgedest = h_uip_edgelist;
	int32_t * actiondest = h_ip_actionslist;
	for(uint64_t j = 0, k = 0; j < h_ui_edges || k < h_ui_actions; j += EDGEBATCHSIZE, k += ACTIONBATCHSIZE) {
		if(j < h_ui_edges) {
			uint32_t generate = (h_ui_edges - j > EDGEBATCHSIZE ? EDGEBATCHSIZE : h_ui_edges - j);
			cudaRMATEdges<<<h_ui_blocks, h_ui_threads>>>(d_uip_randvals, h_ui_scale, generate / (h_ui_blocks * h_ui_threads), h_ui_blocks * h_ui_threads, 0.55, 0.1, 0.1, 0.25, d_uip_edgelist);
			hipMemcpy(edgedest, d_uip_edgelist, generate * 4 * sizeof(uint32_t), hipMemcpyDeviceToHost);
			edgedest += EDGEBATCHSIZE;
		}
		if(k < h_ui_actions) {
			uint32_t generate = (h_ui_actions - k > ACTIONBATCHSIZE ? ACTIONBATCHSIZE : h_ui_actions - k);
			cudaRMATEdges<<<h_ui_blocks, h_ui_threads>>>(d_uip_randvals, h_ui_scale, generate / (h_ui_blocks * h_ui_threads), h_ui_blocks * h_ui_threads, 0.55, 0.1, 0.1, 0.25, tempactions);
			cudaGenerateActions<<<h_ui_blocks / 8, h_ui_threads>>>(d_uip_randvals, h_ui_edges, generate, h_ui_blocks * h_ui_threads / 8, 0.0625, d_uip_edgelist, tempactions, d_ip_actionslist, d_gep_error); 
			hipMemcpy(actiondest, d_ip_actionslist, generate * 4 * sizeof(uint32_t), hipMemcpyDeviceToHost);
			actiondest += ACTIONBATCHSIZE; 
		}
	}

	hipFree(d_uip_edgelist);
	hipFree(d_ip_actionslist);
	hipFree(tempactions);
	
	hipDeviceSynchronize();
	printf("\ncudaRMATEdges() %f", tic_sincelast());

	if(h_s_outfile != NULL) {
		FILE * fp;
		fp = fopen(h_s_outfile, "w+"); 

		if(fp == NULL) {
			fprintf(stderr, "\nERROR: Could not open output file.\n");
			exit(-1);
		}

		uint32_t written = 0;
		written += fwrite(&h_ui_scale, sizeof(uint32_t), 1, fp);
		written += fwrite(&h_ui_edgefactor, sizeof(uint32_t), 1, fp);

		if(written != 2) {
			fprintf(stderr, "\nERROR: Opened output file, but could not write to it.\n");
			exit(-1);
		}

		written = fwrite(h_uip_edgelist, sizeof(uint32_t), 4 * h_ui_edges, fp); 

		if(written != 4 * h_ui_edges) {
			fprintf(stderr, "\nERROR: Opened output file, but could not write to it.\n");
			exit(-1);
		}
		
		fclose(fp);
		hipDeviceSynchronize();
		printf("\nWriteOutputFile() %f", tic_sincelast());
	} 
	
	if(h_s_dimacsoutfile != NULL) {
		FILE * fp;
		fp = fopen(h_s_dimacsoutfile, "w+"); 

		if(fp == NULL) {
			fprintf(stderr, "\nERROR: Could not open output file.\n");
			exit(-1);
		}

		fprintf(fp, "c graph generated by CUDARMAT\n");
		fprintf(fp, "p sp %d %d\n", h_ui_vertices, 2 * h_ui_edges);

		uint32_t j;
		for(j = 0; j < h_ui_edges * 4; j += 2)
			fprintf(fp, "a %d %d 1", h_uip_edgelist[j], h_uip_edgelist[j+1]);
		
		fclose(fp);
		hipDeviceSynchronize();
		printf("\nWriteDimacsOutputFile() %f", tic_sincelast());
	}

	if(h_s_stinger_outfile != NULL) {
		FILE * fp;
		fp = fopen(h_s_stinger_outfile, "w+"); 

		if(fp == NULL) {
			fprintf(stderr, "\nERROR: Could not open output file.\n");
			exit(-1);
		}

		uint32_t written = 0;
		int64_t v64 = h_ui_vertices;
		int64_t e64 = h_ui_edges * 2;     
		int64_t ec = 0x1234ABCD;
		written += fwrite(&ec, sizeof(int64_t), 1, fp);
		written += fwrite(&v64, sizeof(int64_t), 1, fp);
		written += fwrite(&e64, sizeof(int64_t), 1, fp);

		if(written != 3) {
			fprintf(stderr, "\nERROR: Opened output file, but could not write to it.\n");
			exit(-1);
		}

		qsort(h_uip_edgelist, 2 * h_ui_edges, 2 * sizeof(uint32_t), hostCompareEdges);

		int64_t * off = (int64_t *)xcalloc((h_ui_vertices + 1), sizeof(int64_t));
		int64_t * ind = (int64_t *)xmalloc(h_ui_edges * 2 * sizeof(int64_t));
		int64_t * weight = (int64_t *)xmalloc(h_ui_edges * 2 * sizeof(int64_t));

		off += 1;
		uint32_t j, k = 0;
		for(j = 0; j < h_ui_edges * 4; j += 2) {
			off[h_uip_edgelist[j]]++;
			ind[k] = h_uip_edgelist[j+1];
			weight[k] = 1;
			k++;
		}

		for(j = 1; j < h_ui_vertices; ++j)
			off[j] += off[j - 1];
		
		off -= 1;

		written = fwrite(off, sizeof(int64_t), h_ui_vertices + 1, fp);
		written += fwrite(ind, sizeof(int64_t), h_ui_edges * 2, fp);
		written += fwrite(weight, sizeof(int64_t), h_ui_edges * 2, fp);

		if(written != 4 * h_ui_edges + h_ui_vertices + 1) {
			fprintf(stderr, "\nERROR: Opened output file, but could not write to it.\n");
			exit(-1);
		}
		
		free(off);
		free(ind);
		free(weight);
		fclose(fp);
		hipDeviceSynchronize();
		printf("\nWriteSTINGEROutputFile() %f", tic_sincelast());
	}

	if(h_s_stinger_actionsfile != NULL) {
		FILE * fp;
		fp = fopen(h_s_stinger_actionsfile, "w+"); 

		if(fp == NULL) {
			fprintf(stderr, "\nERROR: Could not open output file.\n");
			exit(-1);
		}

		uint32_t written = 0;
		int64_t actions = h_ui_actions * 2;
		uint64_t ec = 0x1234ABCD;
		written += fwrite(&ec, sizeof(int64_t), 1, fp);
		written += fwrite(&actions, sizeof(int64_t), 1, fp);

		if(written != 2) {
			fprintf(stderr, "\nERROR: Opened output file, but could not write to it.\n");
			exit(-1);
		}

		int64_t * act = (int64_t *)xmalloc(sizeof(int64_t) * h_ui_actions * 4);

		for(uint64_t j = 0; j < h_ui_actions * 4; j++) {
			act[j] = h_ip_actionslist[j];
		}

		written = fwrite(act, sizeof(int64_t), h_ui_actions * 4, fp);

		if(written != 4 * h_ui_actions) {
			fprintf(stderr, "\nERROR: Opened output file, but could not write to it.\n");
			exit(-1);
		}

		free(act);
		fclose(fp);
		hipDeviceSynchronize();
		printf("\nWriteSTINGEROutputFile() %f", tic_sincelast());
	}
	free(h_uip_edgelist);
	free(h_ip_actionslist);
}  

__host__ int hostCompareEdges(const void * a, const void * b) {
   uint32_t * e1 = (uint32_t *)a;
   uint32_t * e2 = (uint32_t *)b;

   if(e1[0] == e2[0])
      return e1[1] - e2[1];
   else
      return e1[0] - e2[0];
}

__host__ void hostFreeCudaRand() { 
   hipFree(d_uip_randvals);         
}                              

// HOST DEBUGGING FUNCTIONS
/////////////////////////////
#if(DEBUG)
__host__ void hostDebugTestRand() {
   uint32_t * randvalstest;
   uint32_t bins[32] = {0};
   uint32_t boundaries [] = { 134217728U, 268435456U, 402653184U, 536870912U, 671088640U, 805306368U, 939524096U, 1073741824U,
      1207959552U, 1342177280U, 1476395008U, 1610612736U, 1744830464U, 1879048192U, 2013265920U, 2147483648U, 2281701376U,
      2415919104U, 2550136832U, 2684354560U, 2818572288U, 2952790016U, 3087007744U, 3221225472U, 3355443200U, 3489660928U,
      (uint32_t)3623878656U, (uint32_t)3758096384U, (uint32_t)3892314112U, (uint32_t)4026531840U, (uint32_t)4160749568U, (uint32_t)4294967295U };

   uint32_t numrands = h_ui_blocks * h_ui_threads * TESTRAND;

   randvalstest = (uint32_t *)cudaXmalloc(numrands * sizeof(uint32_t));

   cudaDebugRand <<<h_ui_blocks, h_ui_threads>>>(randvalstest, d_uip_randvals);

   uint32_t * results = (uint32_t *) xmalloc(numrands * sizeof(uint32_t));

   hipMemcpy(results, randvalstest, numrands * sizeof(uint32_t), hipMemcpyDeviceToHost);

   int i,j;
   //int duplicates = 0;

   for(i = 0; i < numrands; ++i) {
      //printf("%u\n", results[i]);

      for(j = 0; j < 32; ++j) {
         if(results[i] < boundaries[j]) {
            bins[j]++;
            break;
         }
      }

      //for(j = i+1; j < numrands; ++j) {
      // if(results[i] == results[j])
      //    duplicates++;
      //}
   }

   free(results);

   printf("\n\n***BEGIN BINS***\n\n");
   for(i = 0; i < 32; ++i) {
      printf("%u\n", bins[i]);
   }

   //printf("\n***DUPLICATES %d***\n", duplicates);

   hipFree(randvalstest);
}

#endif

// Device Functions
//////////////////////////////
__device__ uint32_t cudaRand(uint32_t * randVal) {
  (*randVal) = ((*randVal) * RANDMULT + RANDINCREMENT);
  return *randVal;
}

__global__ void cudaRMATEdges(uint32_t * randVals, uint32_t SCALE, uint32_t edgesPerThread, uint32_t numthreads, 
                                 float pA, float pB, float pC, float pD, uint32_t * edgeArray) {
   __shared__ uint32_t aboutToWrite[MAXBLOCKSIZE];
   uint32_t thread_id = (blockIdx.x * blockDim.x + threadIdx.x);
   int swap = thread_id % 2 == 0 ? 1 : -1;
   uint32_t myRand = randVals[thread_id];  

   float A, B, C, D;

   uint32_t iteration; 
   uint32_t step = numthreads * 4;
   uint32_t stop = numthreads * 4 * edgesPerThread;
   for(iteration = 0; iteration < stop; iteration += step) {
      A = pA;
      B = pB;
      C = pC;
      D = pD;

      uint32_t i = 0;
      uint32_t j = 0;
      uint32_t curBit = ((uint32_t) 1) << (SCALE - 1);

      while(1) {
         const float rand = ((float)cudaRand(&myRand)) / (4294967295.0f);

         if(rand > A) {
            if(rand <= A + B)
               j |= curBit;
            else if (rand <= A + B + C)
               i |= curBit;
            else {
               j |= curBit;
               i |= curBit;
            }
         }

         if(1 == curBit)
            break;

         A *= (0.95 + (((float)cudaRand(&myRand)) / (42949672950.0f)));
         B *= (0.95 + (((float)cudaRand(&myRand)) / (42949672950.0f)));
         C *= (0.95 + (((float)cudaRand(&myRand)) / (42949672950.0f)));
         D *= (0.95 + (((float)cudaRand(&myRand)) / (42949672950.0f)));
         
         const float norm = 1.0 / (A + B + C + D);
         A *= norm;
         B *= norm;
         C *= norm;
         D = 1.0 - (A + B + C);

         curBit >>= 1;
      }

      if(swap == 1) {
         aboutToWrite[threadIdx.x] = i;
      } else {
         aboutToWrite[threadIdx.x] = j;
      }
     
      if(swap == 1) { 
         if(aboutToWrite[threadIdx.x + swap] == j) {
            j ^= 1;
         }
      } else {
         if(aboutToWrite[threadIdx.x + swap] == i)  {
            i ^= 1;                            
         }
      }

      __syncthreads();

      uint32_t index = thread_id + iteration;
      edgeArray[index] = i;
      index += numthreads;
      edgeArray[index] = j;
      index += numthreads;
      edgeArray[index + swap] = j;
      index += numthreads;
      edgeArray[index + swap] = i;
   }
}

__global__ void cudaGenerateActions(uint32_t * randVals, uint32_t edges, uint32_t actions, uint32_t numthreads, 
                                    float pDelete, uint32_t * edgeArray, uint32_t * generatedEdges, int32_t * actionsEdgeArray, graphError_t * error) {
   uint32_t thread_id = (blockIdx.x * blockDim.x + threadIdx.x);
   uint32_t threadIdx4 = threadIdx.x * 4;

   __shared__ int32_t sharedActions[2048];

   uint32_t myRand         = randVals[thread_id];  
   uint32_t original_del   = thread_id * 2;
   uint32_t new_del        = original_del;
   uint32_t new_ins        = original_del;
   uint32_t stop           = 4 * actions;
   uint32_t step           = 4 * numthreads;
   uint32_t index = thread_id;
   for(; index < stop; index += step) {
      const float rand = ((float)cudaRand(&myRand)) / (4294967295.0f);
      if(rand >= pDelete) {
         sharedActions[threadIdx4]   = generatedEdges[new_ins];
         sharedActions[threadIdx4 +1] = generatedEdges[new_ins+1];
         new_ins += step;
      } else {
         if(original_del < edges * 4) {
            sharedActions[threadIdx4]   = -edgeArray[original_del];
            sharedActions[threadIdx4 +1] = -edgeArray[original_del+1];
            original_del += step;
         } else if(new_del < new_ins) {
            sharedActions[threadIdx4]   = -generatedEdges[new_del];
            sharedActions[threadIdx4 +1] = -generatedEdges[new_del+1];
            new_del += step;
         } else {
            // Deletes caught up to insertions
            // if you are near a window, check for flying pigs
            *error = DELETIONS_OVERRAN_INSERTIONS;
         }
      }
      
      //reverse edges
      sharedActions[threadIdx4 + 2] = sharedActions[threadIdx4 + 1];
      sharedActions[threadIdx4 + 3] = sharedActions[threadIdx4];
      __syncthreads();

      actionsEdgeArray[index]                   = sharedActions[threadIdx.x];
      actionsEdgeArray[index + blockDim.x]      = sharedActions[threadIdx.x + blockDim.x];
      actionsEdgeArray[index + blockDim.x * 2]  = sharedActions[threadIdx.x + blockDim.x * 2];
      actionsEdgeArray[index + blockDim.x * 3]  = sharedActions[threadIdx.x + blockDim.x * 3]; 
   }
}


// DEVICE DEBUGGING FUNCTIONS
//////////////////////////////
__global__ void cudaDebugRand(uint32_t * randOut, uint32_t * randVals) {
   int i;
   int thread_id = (blockIdx.x * blockDim.x + threadIdx.x); 
   int thread_offset = thread_id * TESTRAND; 

   for(i = 0; i < TESTRAND; ++i) {
      randOut[i + thread_offset] = cudaRand(randVals + thread_id);
   }
}

__global__ void cudaDebugEdgeList(uint32_t vertices, uint32_t * edgeList, uint32_t size) {
   uint32_t i = 0;
   for(i = 0; i < size; ++i) {
      if(edgeList[i] > vertices)
         edgeList[i] = 0xFFFFFFFF - i;
   }
}
